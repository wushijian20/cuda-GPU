#include <stdio.h>
#include "../tools/common.cuh"

int main(void)
{
    float *fpHost_A;
    fpHost_A = (float *)malloc(4);
    memset(fpHost_A, 0, 4);

    float *fpDevice_A;
    hipError_t error = ErrorCheck(hipMalloc((float **)&fpDevice_A, 4), __FILE__, __LINE__);
    hipMemset(fpDevice_A, 0, 4);

    ErrorCheck(hipMemcpy(fpDevice_A, fpHost_A, 4, hipMemcpyHostToDevice), __FILE__, __LINE__);

    free(fpHost_A);
    ErrorCheck(hipFree(fpDevice_A), __FILE__, __LINE__);
    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);

    return 0;
}