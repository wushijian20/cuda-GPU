
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void)
{
    int iDeviceCount = 0;
    hipError_t error = hipGetDeviceCount(&iDeviceCount);


    if (error != hipSuccess || iDeviceCount == 0)
    {
        printf("No CUDA campatable GPU found!\n");
        exit(-1);
    }
    else
    {
        printf("The count of GPUs is %d.\n", iDeviceCount);
    }

    int iDev = 0;
    error = hipSetDevice(iDev);
    if (error != hipSuccess)
    {
        printf("fail to set GPU 0 for computing.\n");
        exit(-1);
    }
    else
    {
        printf("set GPU 0 for computing.\n");
    }

    return 0;

}