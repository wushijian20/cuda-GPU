
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int id =  threadIdx.x + blockIdx.x * blockDim.x;
    printf("Hello World from block %d and thread %d, global id %d\n", bid, tid, id);

}

int main(void)
{
    hello_from_gpu<<<2,4>>>();  // grid_size: 2 (2 blocks in a grid), block_size: 4 (4 thread in a block)
    hipDeviceSynchronize();

    return 0;
}