
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void print_from_gpu(void) {
    printf("Hello World! from thread [%d, %d] \
        from device\n", threadIdx.x, blockIdx.x);
}

int main(void) {
    // cudaDeviceSynchronize();
    printf("Hello World from host!\n");
    print_from_gpu<<<2,1>>>();
    hipDeviceSynchronize();
    return 0;
}